
#include <hip/hip_runtime.h>
#include <vector>   // vector

/*  use this to set the block size of the kernel launches.
    CUDA kernels will be launched with block size blockDimSize by blockDimSize. */
constexpr int blockDimSize = 8;

/*  your job is to write convolveGPU:
    convolveGPU will be called with blockSize blockDimSize x blockDimSize 
    and gridsize ?height/blockDimSize?x?width/blockDimSize?.
    Each thread may have to compute more than one pixel. You will need to stride the computation.
    Look at convolveCPU below for more info.
*/
__global__ void convolveGPU(float const* in, float *out, int width, int height, 
                            float const* kernel, int kernelWidth, int kernelHeight) {
    // Calculate half sizes of the kernel
    const int halfKernelHeight = kernelHeight / 2;
    const int halfKernelWidth = kernelWidth / 2;
    const int redChannel = 2;
    const int greenChannel = 1;
    const int blueChannel = 0;

    // Calculate the global thread indices
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Row index

    // Calculate the total stride (number of threads covering the width and height)
    int strideX = blockDim.x * gridDim.x;
    int strideY = blockDim.y * gridDim.y;

    // Loop over the pixels assigned to this thread with striding
    for (int row = y; row < height; row += strideY) {
        for (int col = x; col < width; col += strideX) {
            // Check if the pixel is within the valid convolution range
            if (row >= halfKernelHeight && row < height - halfKernelHeight &&
                col >= halfKernelWidth && col < width - halfKernelWidth) {

                float redDot = 0.0f, greenDot = 0.0f, blueDot = 0.0f;

                // Apply the convolution kernel
                for (int k = -halfKernelHeight; k <= halfKernelHeight; ++k) {
                    for (int l = -halfKernelWidth; l <= halfKernelWidth; ++l) {
                        int curRow = row + k;
                        int curCol = col + l;

                        // Compute the index for the input image and kernel
                        int imageIdx = (curRow * width + curCol) * 3;
                        int kernelIdx = (k + halfKernelHeight) * kernelWidth + (l + halfKernelWidth);

                        float kernelValue = kernel[kernelIdx];

                        // Accumulate the convolution results for each color channel
                        redDot += in[imageIdx + redChannel] * kernelValue;
                        greenDot += in[imageIdx + greenChannel] * kernelValue;
                        blueDot += in[imageIdx + blueChannel] * kernelValue;
                    }
                }

                // Write the convolution result to the output image
                int outIdx = (row * width + col) * 3;
                out[outIdx + redChannel] = redDot;
                out[outIdx + greenChannel] = greenDot;
                out[outIdx + blueChannel] = blueDot;
            }
        }
    }
}

/* A CPU example of the convolve kernel */
void convolveCPU(float const* in, float *out, int width, int height, float const* kernel, int kernelWidth, int kernelHeight) {
    const int halfKernelHeight = kernelHeight/2;
    const int halfKernelWidth = kernelWidth/2;
    const int redChannel = 2;
    const int greenChannel = 1;
    const int blueChannel = 0;

    /* point-wise loop over the image pixels */
    for (int i = halfKernelHeight; i < height-halfKernelHeight; i += 1) {
        for (int j = halfKernelWidth; j < width-halfKernelWidth; j += 1) {

            /* compute dot product of kernel and sub-image */
            float redDot = 0.0f, greenDot = 0.0f, blueDot = 0.0f;
            for (int k = -halfKernelHeight; k <= halfKernelHeight; k += 1) {
                for (int l = -halfKernelWidth; l <= halfKernelWidth; l += 1) {

                    /* add in[i+k][j+l]*kernel[k][l] to dot product for red, green, and blue */
                    redDot += in[(i+k)*width*3 + (j+l)*3 + redChannel] * kernel[(k+halfKernelHeight)*kernelWidth + (l+halfKernelWidth)];
                    greenDot += in[(i+k)*width*3 + (j+l)*3 + greenChannel] * kernel[(k+halfKernelHeight)*kernelWidth + (l+halfKernelWidth)];
                    blueDot += in[(i+k)*width*3 + (j+l)*3 + blueChannel] * kernel[(k+halfKernelHeight)*kernelWidth + (l+halfKernelWidth)];
                
                }
            }

            /* set out[i][j] to dot product */
            out[i*width*3 + j*3 + redChannel] = redDot;
            out[i*width*3 + j*3 + greenChannel] = greenDot;
            out[i*width*3 + j*3 + blueChannel] = blueDot;

        }
    }
}

/* call the convolveGPU function on each frame */
float convolveFrames(std::vector<float *> const& framesIn, std::vector<float *> &framesOut, int width, int height, float const* kernel, int kernelWidth, int kernelHeight,
    hipStream_t *streams, int numStreams, int gridSizeX, int gridSizeY) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blockSize (blockDimSize, blockDimSize);
    dim3 gridSize (gridSizeX, gridSizeY);

    hipEventRecord(start, 0);
    for (int i = 0; i < framesIn.size(); i += 1) {
        convolveGPU<<<gridSize, blockSize, 0, streams[i % numStreams]>>>(framesIn.at(i), framesOut.at(i), width, height, kernel, kernelWidth, kernelHeight);
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return (elapsed / 1000.0f);
}